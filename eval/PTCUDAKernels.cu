#include "hip/hip_runtime.h"
/*
 * PTCUDAKernels.cu
 *
 *  Created on: Jan 26, 2016
 *      Author: loheac
 */

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
#include "PTSymbolicObjects.h"

__global__ void evaluatePerturbationTheoryTerm( int* dorder, int* fourierSumIndices, double* result ) {

}

__device__ hipDoubleComplex cexpf( hipDoubleComplex z ) {
	hipDoubleComplex result;
	double real = expf( z.x );  // Field x of hipDoubleComplex refers to Re{z}.
								// Field y of hipDoubleComplex refers to Im{z}.
	sincos( z.y, &result.y, &result.x );

	result.x *= real;
	result.y *= real;

	return result;
}

__device__ void evaluateDUdaggerU_1D( int* kBasisSpace, int* kBasisTime,
								      int* xBasisSpaceA, int* xBasisTimeA,
								      int* xBasisSpaceB, int* xBasisTimeB,
								      double* tau, double* mu,
								      int* NX, int* NTAU, hipDoubleComplex* result ) {

	using namespace std;

	double omega = ( 2.0 * (double)(*kBasisTime) + 1.0 ) / (double)(*NTAU);
	double k = ( 2.0 * (double)(*kBasisSpace) ) / (double)(*NX);

	hipDoubleComplex exparg;
	hipDoubleComplex D;
	hipDoubleComplex UdaggerU;
	hipDoubleComplex expresult;

	exparg.x = -(*tau) * ( ( k * k / 2.0 ) - (*mu) );
	exparg.y = -omega;
	expresult = cexpf( exparg );

	D.x = ( expresult.x * ( 1.0 + expresult.x ) + expresult.y * expresult.y ) / ( ( 1.0 + expresult.x ) * ( 1.0 + expresult.x ) + expresult.y * expresult.y );
	D.y = -expresult.x * expresult.y / ( ( 1.0 + expresult.x ) * ( 1.0 + expresult.x ) + expresult.y * expresult.y );

	exparg.x = -k * ( (*xBasisSpaceA) + (*xBasisSpaceB) );
	exparg.y = omega * ( (*xBasisTimeA) + (*xBasisTimeB) );
	UdaggerU = cexpf( exparg );

	(*result).x = D.x * UdaggerU.x - D.y * UdaggerU.y;
	(*result).y = D.x * UdaggerU.y + D.y * UdaggerU.x;
}
